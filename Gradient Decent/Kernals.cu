#include "hip/hip_runtime.h"
// Kernals.cu
// here is where all the device and global functions live

#include "Kernals.hpp"
__device__ void vectorSub(double * v1, double * v2, double * vOut){
    
    *vOut = *v1-*v2;
    *(vOut + 1) = *(v1 + 1) - *(v2 + 1);
    *(vOut + 2) = *(v1 + 2) - *(v2 + 2);
}
__device__ void vectorAdd(double * v1, double * v2, double * vOut) {
    *vOut = *v1 + *v2;
    *(vOut + 1) = *(v1 + 1) + *(v2 + 1);
    *(vOut + 2) = *(v1 + 2) + *(v2 + 2);
}
__device__ void vecScale(double *v, double lambda){
    *v *= lambda;
    *(v+1) *= lambda;
    *(v+2) *= lambda;
}
__device__ void vecAssign(double *out, double *in,double lambda){ // out  = in*lambda
    *out = *in * lambda;
    *(out + 1) = *(in + 1) * lambda;
    *(out + 2) = *(in + 2) * lambda;
}
__device__ void cross(double *a,double *b, double *c) {
    (*c)     = (*(a+1)) * (*(b+2)) - (*(a+2)) * (*(b+1));
    (*(c+1)) = (*(b)) * (*(a+2)) - (*(a)) * (*(b+2));
    (*(c+2)) = (*(a)) * (*(b+1)) - (*(b)) * (*(a+1));
}

__device__ double dot(double *a, double *b) {
     return ((*a) * (*b) + (*(a+1)) * (*(b+1)) + (*(a+2)) * (*(b+2)));
}

__device__ double norm(double *a) {
    return sqrt(dot(a, a));
}

__device__ int sign(double a){
    if (a>0) return 1;
    if (a<0) return -1;
    else return 0;
}


__global__ void areaKernel(double * area, double * vert, unsigned int * facets, unsigned int numFacets){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // do i*3 because we have 3 vertcies per facet
    // do facets[]*3 becasue we have x y and z positions
    double r10[3];
    double r21[3];
    double S[3];

    if (i < numFacets) {
        vectorSub(&vert[facets[i*3+1]*3], &vert[facets[i*3]*3],r10);
        vectorSub(&vert[facets[i*3+2]*3], &vert[facets[i*3+1]*3],r21);    
        cross(r10, r21,S);
        area[i] = norm(S)/2;
        //printf("Thread %d:\tArea %f\n",i,area[i]);
    }
    else {
        area[i] = 0;
    }
}
__global__ void volumeKernel(double * volume, double * vert, unsigned int * facets, unsigned int numFacets){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    double s01[3];
    if (i < numFacets){
        cross(&vert[facets[i*3]*3], &vert[facets[i*3+1]*3],s01);
        volume[i] = abs(dot(s01,&vert[facets[i*3+2]*3]))/6;
    }
    else {
        volume[i] = 0;
    }

}
__global__ void addTree(double* g_idata, double* g_odata){
    //https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

    extern __shared__ double sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x; // get the id of this thread
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    
        //printf("tid: %d\ti:%d\ti + blockDim.x:%d\tg_idata[i]:%f\tg_idata[i + blockDim.x]%f\n",tid,i, i + blockDim.x, g_idata[i] , g_idata[i + blockDim.x]);
        sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];// g_idata[i]; // move the data over
        g_idata[i] = 0;
        g_idata[i + blockDim.x] = 0;
        // printf("tid: %d\ti:%d\ti + blockDim.x:%d\tg_idata[i]:%f\tg_idata[i + blockDim.x]%f\t sdata[tid]: %f\n", tid, i, i + blockDim.x, g_idata[i], g_idata[i + blockDim.x], sdata[tid]);
                                                        //}
    __syncthreads();
            // do reduction in shared mem
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                sdata[tid] += sdata[tid + s];
            }
        __syncthreads();
        }
        __syncthreads();
        
        //     write result for this block to global mem
        if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize> __device__ void warpReduce(volatile double *sdata, unsigned int tid) {
        if (blockSize >=  64) sdata[tid] += sdata[tid + 32];
        if (blockSize >=  32) sdata[tid] += sdata[tid + 16];
        if (blockSize >=  16) sdata[tid] += sdata[tid +  8];
        if (blockSize >=    8) sdata[tid] += sdata[tid +  4];
        if (blockSize >=    4) sdata[tid] += sdata[tid +  2];
        if (blockSize >=    2) sdata[tid] += sdata[tid +  1];
}
template <unsigned int blockSize> __global__ void reduce6(double *g_idata,double *g_odata, unsigned int n) {
    extern __shared__ double sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + tid;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;
    while (i < n) {
        sdata[tid] += g_idata[i] + g_idata[i+blockSize];
        i += gridSize;
    }
    __syncthreads();
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <   64) { sdata[tid] += sdata[tid +   64]; } __syncthreads(); }
    if (tid < 32) warpReduce(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void addWithMultKernel(double *a ,double *b,double lambda, unsigned int size){
    // a += b * lambda
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<size){
        *(a+i) += *(b+i) * lambda;
    }
}

__global__ void areaGradient(double* gradAFacet, unsigned int* facets,double* verts,unsigned int numFacets){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    double S0[3];
    double S1[3];
    double S01[3];
    double S010[3];
    double S011[3];
    if (i<numFacets){
        vectorSub(&verts[facets[i*3+1]*3], &verts[facets[i*3]*3],S0);
        vectorSub(&verts[facets[i*3+2]*3], &verts[facets[i*3+1]*3],S1);
        cross(S0,S1,S01);
        cross(S01,S0,S010);
        cross(S01,S1,S011);
        // each facet has 3 vertices with gradient each, so in total 9 numbers we write them down here;
        
        // or facet i this is the gradent vector for its 0th vertex 

        vecAssign(&gradAFacet[i*9],S011,1.0/(2 * norm(S01)));

        // reuse S0 
        vectorAdd(S011,S010,S0);
        vecAssign(&gradAFacet[i*9 + 3],S0,-1.0/(2 * norm(S01)));

        vecAssign(&gradAFacet[i*9 + 6],S010,1.0/(2 * norm(S01)));
    }

}
__global__ void volumeGradient(double* gradVFacet, unsigned int* facets,double* verts,unsigned int numFacets){
    // TO DO: this can this can be broken up into 3 for even faster computaiton
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    double c[3];
    double s = 1;
    if (i<numFacets){
        cross(&verts[facets[i*3]*3],&verts[facets[i*3+1]*3],c);
        s = sign(dot(c,&verts[facets[i*3+2]*3]));

        cross(&verts[facets[i*3+1]*3],&verts[facets[i*3+2]*3],c);
        vecAssign(&gradVFacet[i*9],c,s/6);

        cross(&verts[facets[i*3+2]*3],&verts[facets[i*3]*3],c);
        vecAssign(&gradVFacet[i*9 + 3],c,s/6);

        cross(&verts[facets[i*3]*3],&verts[facets[i*3+1]*3],c);
        vecAssign(&gradVFacet[i*9 + 6],c,s/6);
    }

}
__global__ void facetToVertex(double* vertexValue, double* facetValue,unsigned int* vertToFacet, unsigned int* vertIndexStart,unsigned int numVert){
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i<numVert){
        //first set to 0
        vertexValue[i*3] = 0;
        vertexValue[i*3 + 1] = 0;
        vertexValue[i*3 + 2] = 0;
        for (int index = vertIndexStart[i]; index < vertIndexStart[i+1]; index++){
            vectorAdd(&vertexValue[i*3],&facetValue[3*vertToFacet[index]],&vertexValue[i*3]);
        }
    }
}

__global__ void projectForce(double* force,double* gradAVert,double* gradVVert,unsigned int numVert){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    double proj[3];
    double scaledGV[3];
    if (i<numVert){
        // project the vector gA - (gA . gV)/(gV . gV) gV
        // first create the (gA . gV)/(gV . gV) gV vector 
        double denom =dot(&gradVVert[i*3],&gradVVert[i*3]);
        if (abs(denom)>0){
            vecAssign(scaledGV,&gradVVert[i*3], dot(&gradAVert[i*3],&gradVVert[i*3])/denom);
        }
        else {
            vecAssign(scaledGV,&gradVVert[i*3], 0);

        }
        // subtract
        vectorSub(&gradAVert[i*3],scaledGV,proj);
        
        // and assgin
        vecAssign(&force[i*3],proj,-1);
        /*
        printf("thread %d \t gradA = [%f,%f,%f]\n",i,gradAVert[i*3],gradAVert[i*3+1],gradAVert[i*3+2]);
        printf("thread %d \t gradV = [%f,%f,%f]\n",i,gradVVert[i*3],gradVVert[i*3+1],gradVVert[i*3+2]);
        printf("thread %d \t scaled GV = [%f,%f,%f]\n",i,scaledGV[0],scaledGV[1],scaledGV[2]);
        printf("thread %d \t f = [%f,%f,%f]\n",i,force[i*3],force[i*3+1],force[i*3+2]);
        */
    }
}