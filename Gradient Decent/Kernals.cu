#include "hip/hip_runtime.h"
// Kernals.cu
// here is where all the device and global functions live

#include "Kernals.hpp"
__device__ void vectorSub(double * v1, double * v2, double * vOut){
    
    *vOut = *v1-*v2;
    *(vOut + 1) = *(v1 + 1) - *(v2 + 1);
    *(vOut + 2) = *(v1 + 2) - *(v2 + 2);
}
__device__ void vectorAdd(double * v1, double * v2, double * vOut) {
    *vOut = *v1 + *v2;
    *(vOut + 1) = *(v1 + 1) + *(v2 + 1);
    *(vOut + 2) = *(v1 + 2) + *(v2 + 2);
}
__device__ void vecScale(double *v, double lambda){
    *v *= lambda;
    *(v+1) *= lambda;
    *(v+2) *= lambda;
}
__device__ void vecAssign(double *out, double *in,double lambda){ // out  = in*lambda
    *out = *in * lambda;
    *(out + 1) = *(in + 1) * lambda;
    *(out + 2) = *(in + 2) * lambda;
}
__device__ void cross(double *a,double *b, double *c) {
    (*c)     = (*(a+1)) * (*(b+2)) - (*(a+2)) * (*(b+1));
    (*(c+1)) = (*(b)) * (*(a+2)) - (*(a)) * (*(b+2));
    (*(c+2)) = (*(a)) * (*(b+1)) - (*(b)) * (*(a+1));
}

__device__ double dot(double *a, double *b) {
     return ((*a) * (*b) + (*(a+1)) * (*(b+1)) + (*(a+2)) * (*(b+2)));
}

__device__ double norm(double *a) {
    return sqrt(dot(a, a));
}

__device__ int sign(double a){
    if (a>0) return 1;
    if (a<0) return -1;
    else return 0;
}


__global__ void areaKernel(double * area, double * vert, unsigned int * facets, unsigned int numFacets){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // do i*3 because we have 3 vertcies per facet
    // do facets[]*3 becasue we have x y and z positions
    double r10[3];
    double r21[3];
    double S[3];

    if (i < numFacets) {
        vectorSub(&vert[facets[i*3+1]*3], &vert[facets[i*3]*3],r10);
        vectorSub(&vert[facets[i*3+2]*3], &vert[facets[i*3+1]*3],r21);    
        cross(r10, r21,S);
        area[i] = norm(S)/2;
        //printf("Thread %d:\tArea %f\n",i,area[i]);
    }
    else {
        area[i] = 0;
    }
}
__global__ void volumeKernel(double * volume, double * vert, unsigned int * facets, unsigned int numFacets){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    double s01[3];
    if (i < numFacets){
        cross(&vert[facets[i*3]*3], &vert[facets[i*3+1]*3],s01);
        volume[i] = abs(dot(s01,&vert[facets[i*3+2]*3]))/6;
    }
    else {
        volume[i] = 0;
    }

}
__global__ void addTree(double* g_idata, double* g_odata){
    //https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

    extern __shared__ double sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x; // get the id of this thread
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    
        //printf("tid: %d\ti:%d\ti + blockDim.x:%d\tg_idata[i]:%f\tg_idata[i + blockDim.x]%f\n",tid,i, i + blockDim.x, g_idata[i] , g_idata[i + blockDim.x]);
        sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];// g_idata[i]; // move the data over
        g_idata[i] = 0;
        g_idata[i + blockDim.x] = 0;
        // printf("tid: %d\ti:%d\ti + blockDim.x:%d\tg_idata[i]:%f\tg_idata[i + blockDim.x]%f\t sdata[tid]: %f\n", tid, i, i + blockDim.x, g_idata[i], g_idata[i + blockDim.x], sdata[tid]);
                                                        //}
    __syncthreads();
            // do reduction in shared mem
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                sdata[tid] += sdata[tid + s];
            }
        __syncthreads();
        }
        __syncthreads();
        
        //     write result for this block to global mem
        if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

// template <unsigned int blockSize> __device__ void warpReduce(volatile double *sdata, unsigned int tid) {
//         if (blockSize >=  64) sdata[tid] += sdata[tid + 32];
//         if (blockSize >=  32) sdata[tid] += sdata[tid + 16];
//         if (blockSize >=  16) sdata[tid] += sdata[tid +  8];
//         if (blockSize >=    8) sdata[tid] += sdata[tid +  4];
//         if (blockSize >=    4) sdata[tid] += sdata[tid +  2];
//         if (blockSize >=    2) sdata[tid] += sdata[tid +  1];
// }
// template <unsigned int blockSize> __global__ void reduce6(double *g_idata,double *g_odata, unsigned int n) {
//     extern __shared__ double sdata[];
//     unsigned int tid = threadIdx.x;
//     unsigned int i = blockIdx.x*(blockSize*2) + tid;
//     unsigned int gridSize = blockSize*2*gridDim.x;
//     sdata[tid] = 0;
//     while (i < n) {
//         sdata[tid] += g_idata[i] + g_idata[i+blockSize];
//         i += gridSize;
//     }
//     __syncthreads();
//     if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
//     if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
//     if (blockSize >= 128) { if (tid <   64) { sdata[tid] += sdata[tid +   64]; } __syncthreads(); }
//     if (tid < 32) warpReduce(sdata, tid);
//     if (tid == 0) g_odata[blockIdx.x] = sdata[0];
// }

__global__ void addWithMultKernel(double *a ,double *b,double lambda, unsigned int size){
    // a += b * lambda
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<size){
        *(a+i) += *(b+i) * lambda;
    }
}

__global__ void areaGradient(double* gradAFacet, unsigned int* facets,double* verts,unsigned int numFacets){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    double S0[3];
    double S1[3];
    double S01[3];
    double S010[3];
    double S011[3];
    if (i<numFacets){
        vectorSub(&verts[facets[i*3+1]*3], &verts[facets[i*3]*3],S0);
        vectorSub(&verts[facets[i*3+2]*3], &verts[facets[i*3+1]*3],S1);
        cross(S0,S1,S01);
        cross(S01,S0,S010);
        cross(S01,S1,S011);
        // each facet has 3 vertices with gradient each, so in total 9 numbers we write them down here;
        
        // or facet i this is the gradent vector for its 0th vertex 

        vecAssign(&gradAFacet[i*9],S011,1.0/(2 * norm(S01)));

        // reuse S0 
        vectorAdd(S011,S010,S0);
        vecAssign(&gradAFacet[i*9 + 3],S0,-1.0/(2 * norm(S01)));

        vecAssign(&gradAFacet[i*9 + 6],S010,1.0/(2 * norm(S01)));
    }

}
__global__ void volumeGradient(double* gradVFacet, unsigned int* facets,double* verts,unsigned int numFacets){
    // TO DO: this can this can be broken up into 3 for even faster computaiton
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    double c[3];
    double s = 1;
    if (i<numFacets){
        cross(&verts[facets[i*3]*3],&verts[facets[i*3+1]*3],c);
        s = sign(dot(c,&verts[facets[i*3+2]*3]));

        cross(&verts[facets[i*3+1]*3],&verts[facets[i*3+2]*3],c);
        vecAssign(&gradVFacet[i*9],c,s/6);

        cross(&verts[facets[i*3+2]*3],&verts[facets[i*3]*3],c);
        vecAssign(&gradVFacet[i*9 + 3],c,s/6);

        cross(&verts[facets[i*3]*3],&verts[facets[i*3+1]*3],c);
        vecAssign(&gradVFacet[i*9 + 6],c,s/6);
    }

}
__global__ void facetToVertex(double* vertexValue, double* facetValue,unsigned int* vertToFacet, unsigned int* vertIndexStart,unsigned int numVert){
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i<numVert){
        //first set to 0
        vertexValue[i*3] = 0;
        vertexValue[i*3 + 1] = 0;
        vertexValue[i*3 + 2] = 0;
        for (int index = vertIndexStart[i]; index < vertIndexStart[i+1]; index++){
            vectorAdd(&vertexValue[i*3],&facetValue[3*vertToFacet[index]],&vertexValue[i*3]);
            //printf("vertex %d gets [%f,%f,%f]\n",i,facetValue[3*vertToFacet[index]],facetValue[3*vertToFacet[index]+1],facetValue[3*vertToFacet[index]+2]);
        }
    }
}

__global__ void projectForce(double* force,double* gradAVert,double* gradVVert,double scale,unsigned int numEle){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<numEle){
        force[i] = - (gradAVert[i] - scale * gradVVert[i]);
    }
}

__global__ void elementMultiply(double* v1, double* v2, double* out, unsigned int size){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<size){
        out[i] = v1[i]*v2[i];
        //printf("Thread %d: out value %f\n",i,out[i]);
    }
}



double sum_of_elements(hipError_t cudaStatus,double* vec,unsigned int size,unsigned int bufferedSize,unsigned int blockSize){

    double out;

    // do the reduction each step sums blockSize*2 number of elements
    unsigned int numberOfBlocks = ceil(size / (float) blockSize / 2.0);
    // printf("AddTree with %d blocks,  of blocks size %d, for %d total elements\n",numberOfBlocks,blockSize,_bufferedSize);
    
    addTree<<<numberOfBlocks, blockSize, bufferedSize / 2 * sizeof(double) >>> (vec, vec);


    if (numberOfBlocks>1){
        for (int i = numberOfBlocks; i > 1; i /= (blockSize * 2)) {
            addTree<<<ceil((float)numberOfBlocks/ (blockSize * 2)), blockSize, ceil((float)size / 2)* sizeof(double) >>> (vec, vec);
        } 
    }
    cuda_sync_and_check(cudaStatus,"sum of elements");

    // copy the 0th element out of the vector now that it contains the sum
    cudaStatus = hipMemcpy(&out, vec,sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! area\n");
    throw;
    }

    return out;

}

void cuda_sync_and_check(hipError_t cudaStatus, const char * caller){
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s. From %s\n", hipGetErrorString(cudaStatus),caller);
        throw "Kernel Launch Failure";
    }
    // check that the kernal didn't throw an error
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error %s after launching Kernel %s!\n", hipGetErrorString(cudaStatus),caller);
        throw "Kernel Failure";
    }

}
double dotProduct(hipError_t cudaStatus,double * v1, double * v2, double * scratch, unsigned int size, unsigned int blockSize){

    // first multiply
    unsigned int numberOfBlocks = ceil(size / (float) blockSize);

    elementMultiply<<<numberOfBlocks,blockSize>>>(v1,v2, scratch,size);
    cuda_sync_and_check(cudaStatus,"Element Multiply");
    unsigned int bufferedSize = ceil(size/(2.0*blockSize))*2 *blockSize;
    //now sum
    double out = sum_of_elements(cudaStatus,scratch,size, bufferedSize,blockSize);
    
    // clear the scratch
    hipMemset(scratch,0,sizeof(double)*bufferedSize);
    cuda_sync_and_check(cudaStatus,"dotProduct");

    return out;


}