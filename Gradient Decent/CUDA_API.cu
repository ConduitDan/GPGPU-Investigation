#include "hip/hip_runtime.h"
// Kernals.cu
// here is where all the device and global functions live

#include "CUDA_API.hpp"


// make this thing a singleton

CUDA::CUDA():DeviceAPI(256){
    _cudaStatus = hipSetDevice(0);
    if (_cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
}
CUDA::CUDA(int blockSizeIn):DeviceAPI(blockSizeIn){
	printf("Setting up GPU with blocksize %d\n",blockSizeIn);
    _cudaStatus = hipSetDevice(0);
    if (_cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
}

CUDA::~CUDA(){
	hipDeviceReset();
}

void CUDA::allocate(void** ptr, unsigned int size){


    _cudaStatus = hipMalloc((void**)ptr,(int) size);
    if (_cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!%s\n",hipGetErrorString(_cudaStatus));
    }
}
void CUDA::deallocate(void* devicePointer){
	if (devicePointer) {
		_cudaStatus = hipFree(devicePointer);
	}
	if (_cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFree failed! %s\n",hipGetErrorString(_cudaStatus));
		throw "oops";
    }

}


void CUDA::copy_to_device(void* devicePointer, void* hostPointer, unsigned int size){
    _cudaStatus = hipMemcpy(devicePointer, hostPointer, size, hipMemcpyHostToDevice);
    if (_cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! %s\n",hipGetErrorString(_cudaStatus));
		throw "oops";
    }
}

void CUDA::copy_to_host(void * hostPointer, void * devicepointer, unsigned int size){
    _cudaStatus = hipMemcpy(hostPointer, devicepointer, size, hipMemcpyDeviceToHost);
    if (_cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! %s\n",hipGetErrorString(_cudaStatus));
		throw "oops";
    }
}


void CUDA::project_force(UniqueDevicePtr<double>* force,UniqueDevicePtr<double>* gradAVert,UniqueDevicePtr<double>* gradVVert, double scale,unsigned int size){
    unsigned int numberOfBlocks = ceil(size / (float) blockSize);
    projectForce<<<numberOfBlocks, blockSize>>>((double*)force->get(),(double*)gradAVert->get(),(double*)gradVVert->get(),scale,size);
    cuda_sync_and_check("project to force");

}
void CUDA::facet_to_vertex(UniqueDevicePtr<double>* vertexValue, UniqueDevicePtr<double>* facetValue,UniqueDevicePtr<unsigned int>* vertToFacet, UniqueDevicePtr<unsigned int>* vertIndexStart,unsigned int numVert){
    unsigned int numberOfBlocks = ceil(numVert / (float) blockSize);
	dim3 numberOfBlocks3D( numberOfBlocks, 1, 1 );
	dim3 blockSize3D( blockSize, 3, 1 );
    facetToVertex<<<numberOfBlocks3D, blockSize3D>>>((double*)vertexValue->get(),\
												 (double*)facetValue->get(),\
												 (unsigned int*)vertToFacet->get(),\
												 (unsigned int*)vertIndexStart->get(),numVert);
    cuda_sync_and_check("facet_to_vertex");

}



void CUDA::area_gradient(UniqueDevicePtr<double>* gradAFacet,UniqueDevicePtr<unsigned int>* facets,UniqueDevicePtr<double>* vert,unsigned int numFacets){
    unsigned int numberOfBlocks = ceil(numFacets / (float) blockSize);
    areaGradient<<<numberOfBlocks, blockSize>>>((double*)gradAFacet->get(),(unsigned int*)facets->get(),(double*)vert->get(),numFacets);
    cuda_sync_and_check("GradA");


}

void CUDA::volume_gradient(UniqueDevicePtr<double>* gradVFacet,UniqueDevicePtr<unsigned int>* facets,UniqueDevicePtr<double>* vert,unsigned int numFacets){
    unsigned int numberOfBlocks = ceil(numFacets / (float) blockSize);
    volumeGradient<<<numberOfBlocks, blockSize>>>((double*)gradVFacet->get(), (unsigned int*)facets->get(), (double*)vert->get(), numFacets);
    cuda_sync_and_check("GradV");

}
void CUDA::area_gradient2(UniqueDevicePtr<double>* gradAVert,UniqueDevicePtr<unsigned int>* facets,UniqueDevicePtr<double>* vert,unsigned int numFacets,unsigned int numVert){
	// mem set and call kernal
    hipMemset(gradAVert->get(),0.0,sizeof(double)*numVert*3);
	cuda_sync_and_check("memset");
	unsigned int numberOfBlocks = ceil(numFacets / (float) blockSize);
    areaGradient2<<<numberOfBlocks, blockSize>>>((double*)gradAVert->get(), (unsigned int*)facets->get(), (double*)vert->get(),numFacets);
    cuda_sync_and_check("GradA");
//(double* gradVVert, unsigned int* facets,double* verts,unsigned int numFacets)
}
void CUDA::volume_gradient2(UniqueDevicePtr<double>* gradVVert,UniqueDevicePtr<unsigned int>* facets,UniqueDevicePtr<double>* vert,unsigned int numFacets,unsigned int numVert){
    hipMemset(gradVVert->get(),0.0,sizeof(double)*numVert*3);
	cuda_sync_and_check("memset");

    unsigned int numberOfBlocks = ceil(numFacets / (float) blockSize);
    volumeGradient2<<<numberOfBlocks, blockSize>>>((double*)gradVVert->get(), (unsigned int*)facets->get(), (double*)vert->get(),numFacets);
    cuda_sync_and_check("GradV");

}
void CUDA::area_gradient3(UniqueDevicePtr<double>* gradAVert,UniqueDevicePtr<unsigned int>* facets,UniqueDevicePtr<double>* vert,unsigned int numFacets,unsigned int numVert){
	// mem set and call kernal
    hipMemset(gradAVert->get(),0.0,sizeof(double)*numVert*3);
	cuda_sync_and_check("memset");

	unsigned int numberOfBlocks = ceil(numFacets / (float) blockSize);

	dim3 numberOfBlocks3D( numberOfBlocks, 1, 1 );
	dim3 blockSize3D( blockSize, 3, 1 );

    areaGradient3<<<numberOfBlocks3D, blockSize3D,15*blockSize*sizeof(double)>>>((double*)gradAVert->get(), (unsigned int*)facets->get(), (double*)vert->get(),numFacets);
    cuda_sync_and_check("GradA");
//(double* gradVVert, unsigned int* facets,double* verts,unsigned int numFacets)
}
void CUDA::volume_gradient3(UniqueDevicePtr<double>* gradVVert,UniqueDevicePtr<unsigned int>* facets,UniqueDevicePtr<double>* vert,unsigned int numFacets,unsigned int numVert){
    
	



	// int tmpBlockSize = ceil((float)blockSize/9);
	// // find the closest multiple of 32 
	// tmpBlockSize = ceil((float)tmpBlockSize/32)*32; 


    // unsigned int numberOfBlocks = floor(numFacets / (float) tmpBlockSize);
	// tmpBlockSize = 1;
	// numberOfBlocks = numFacets;
	// dim3 blockSize3D( tmpBlockSize, 3, 3 );
	// unsigned int numberOfBlocks = floor(numFacets / (float) blockSize);
	// dim3 blockSize3D( blockSize, 3, 3 );
	// dim3 numberOfBlocks3D( numberOfBlocks, 1);
	
	hipMemset(gradVVert->get(),0.0,sizeof(double)*numVert*3);
	cuda_sync_and_check("memset");

	unsigned int tmpBlockSize = 32;
    unsigned int numberOfBlocks = ceil(numFacets / (float) tmpBlockSize);

	dim3 numberOfBlocks3D( numberOfBlocks, 1, 1 );
	dim3 blockSize3D( tmpBlockSize, 3, 3 );
//	printf("calling volGrad3 with %d blocks of size %d X %d X %d \n",numberOfBlocks,blockSize,3,3);

    volumeGradient3<<<numberOfBlocks3D, blockSize3D,blockSize*sizeof(double)>>>((double*)gradVVert->get(), (unsigned int*)facets->get(), (double*)vert->get(),numFacets);
    cuda_sync_and_check("GradV");

}


double CUDA::sum_of_elements(UniqueDevicePtr<double>* vec,unsigned int size,unsigned int bufferedSize){

    double out;
	
    // do the reduction each step sums blockSize*2 number of elements
    unsigned int numberOfBlocks = ceil(size / (float) blockSize / 2.0);
    //printf("AddTree with %d blocks,  of blocks size %d, for %d total elements\n",numberOfBlocks,blockSize,bufferedSize);
    
    addTree<<<numberOfBlocks, blockSize, blockSize  * sizeof(double) >>> ((double*)vec->get(), (double*)vec->get());
    cuda_sync_and_check("sum of elements");


    if (numberOfBlocks>1){
        for (int i = numberOfBlocks; i > 1; i /= (blockSize * 2)) {
            addTree<<<ceil((float)numberOfBlocks/ (blockSize * 2)), blockSize, blockSize * sizeof(double)>>> ((double*)vec->get(), (double*)vec->get());
            cuda_sync_and_check("sum of elements");
        } 
    }

    // copy the 0th element out of the vector now that it contains the sum
    copy_to_host(&out, vec->get(),sizeof(double));
  

    return out;

}

void CUDA::cuda_sync_and_check(const char * caller){
    _cudaStatus = hipGetLastError();
    if (_cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s. From %s\n", hipGetErrorString(_cudaStatus),caller);
        throw "Kernel Launch Failure";
    }
    // check that the kernal didn't throw an error
    _cudaStatus = hipDeviceSynchronize();
    if (_cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error %s after launching Kernel %s!\n", hipGetErrorString(_cudaStatus),caller);
        throw "Kernel Failure";
    }

}



double CUDA::dotProduct(UniqueDevicePtr<double>* v1, UniqueDevicePtr<double>* v2, UniqueDevicePtr<double>* scratch, unsigned int size){

    // first multiply
    unsigned int numberOfBlocks = ceil(size / (float) blockSize);

    elementMultiply<<<numberOfBlocks,blockSize>>>((double*)v1->get(),(double*)v2->get(), (double*)scratch->get(),size);
    cuda_sync_and_check("Element Multiply");
    unsigned int bufferedSize = ceil(size/(2.0*blockSize))*2 *blockSize;
    //now sum
    double out = sum_of_elements(scratch,size, bufferedSize);
	cuda_sync_and_check("sum_of_elments");

    // clear the scratch
    hipMemset(scratch->get(),0.0,sizeof(double)*bufferedSize);
	cuda_sync_and_check("memset");

    return out;


}


void CUDA::area(UniqueDevicePtr<double>* area, UniqueDevicePtr<double>* vert, UniqueDevicePtr<unsigned int>* facets, unsigned int numFacets){
    unsigned int numberOfBlocks = ceil(numFacets / (float) blockSize);
    areaKernel<<<numberOfBlocks,blockSize>>>((double*)area->get(),(double*) vert->get(), (unsigned int*) facets->get(), numFacets);
    cuda_sync_and_check("area");

}
void CUDA::volume(UniqueDevicePtr<double>* volume, UniqueDevicePtr<double>* vert, UniqueDevicePtr<unsigned int>* facets, unsigned int numFacets){
	
	cuda_sync_and_check("inb4 volume");
    unsigned int numberOfBlocks = ceil(numFacets / (float) blockSize);
    volumeKernel<<<numberOfBlocks,blockSize>>>((double*)volume->get(),(double*) vert->get(), (unsigned int*) facets->get(), numFacets);
    cuda_sync_and_check("volume");

}
void CUDA::add_with_mult(UniqueDevicePtr<double>* a,UniqueDevicePtr<double>* b, double lambda, unsigned int size){
    unsigned int numberOfBlocks = ceil(size / (float) blockSize);
    addWithMultKernel<<<numberOfBlocks,blockSize>>>((double*)a->get(),(double*)b->get(),lambda,size);
}

double CUDA::getGPUElement(double * vec, unsigned int index){
	double out;
	_cudaStatus = hipMemcpy(&out, vec + index ,sizeof(double), hipMemcpyDeviceToHost);
    if (_cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! area\n");
    throw;
    }

    return out;
}

unsigned int CUDA::getGPUElement(unsigned int * vec, unsigned int index){
	unsigned int out;
	_cudaStatus = hipMemcpy(&out, vec + index ,sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (_cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! area\n");
    throw;
    }

    return out;
}
#include "kernalfile.hpp" 

__device__ double atomic_Add(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ void vectorSub(double * v1, double * v2, double * vOut){
    
    *vOut = *v1-*v2;
    *(vOut + 1) = *(v1 + 1) - *(v2 + 1);
    *(vOut + 2) = *(v1 + 2) - *(v2 + 2);
}
__device__ void vectorAdd(double * v1, double * v2, double * vOut) {
    *vOut = *v1 + *v2;
    *(vOut + 1) = *(v1 + 1) + *(v2 + 1);
    *(vOut + 2) = *(v1 + 2) + *(v2 + 2);
}
__device__ void vecScale(double *v, double lambda){
    *v *= lambda;
    *(v+1) *= lambda;
    *(v+2) *= lambda;
}
__device__ void vecAssign(double *out, double *in,double lambda){ // out  = in*lambda
    *out = *in * lambda;
    *(out + 1) = *(in + 1) * lambda;
    *(out + 2) = *(in + 2) * lambda;
}
__device__ void cross(double *a,double *b, double *c) {
    (*c)     = (*(a+1)) * (*(b+2)) - (*(a+2)) * (*(b+1));
    (*(c+1)) = (*(b)) * (*(a+2)) - (*(a)) * (*(b+2));
    (*(c+2)) = (*(a)) * (*(b+1)) - (*(b)) * (*(a+1));
}
__device__ void cross3(double *a,double *b, double *c,int k) {
	if (k == 0){
    	(*c)     = (*(a+1)) * (*(b+2)) - (*(a+2)) * (*(b+1));
	}
	if (k == 1){
    	(*(c+1)) = (*(b)) * (*(a+2)) - (*(a)) * (*(b+2));
	}
	if (k == 2){
    	(*(c+2)) = (*(a)) * (*(b+1)) - (*(b)) * (*(a+1));
	}
}

__device__ double cross3oneComp(double *a,double *b,int k) {
	if (k == 0){
    	//return (*(a+1)) * (*(b+2)) - (*(a+2)) * (*(b+1));
		return a[1]*b[2]-a[2]*b[1];
		
	}
	if (k == 1){
    	//return (*(b)) * (*(a+2)) - (*(a)) * (*(b+2));
		return a[2]*b[0]-a[0]*b[2];
	}
	if (k == 2){
    	//return (*(a)) * (*(b+1)) - (*(b)) * (*(a+1));
		return a[0]*b[1]-a[1]*b[0];
	}
	
	return 0;
}

__device__ double dot(double *a, double *b) {
     return ((*a) * (*b) + (*(a+1)) * (*(b+1)) + (*(a+2)) * (*(b+2)));
}

__device__ double norm(double *a) {
    return sqrt(dot(a, a));
}

__device__ int sign(double a){
    if (a>0) return 1;
    if (a<0) return -1;
    else return 0;
}



__global__ void areaKernel(double * area, double * vert, unsigned int * facets, unsigned int numFacets){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // do i*3 because we have 3 vertcies per facet
    // do facets[]*3 becasue we have x y and z positions
    double r10[3];
    double r21[3];
    double S[3];

    if (i < numFacets) {
        vectorSub(&vert[facets[i*3+1]*3], &vert[facets[i*3]*3],r10);
        vectorSub(&vert[facets[i*3+2]*3], &vert[facets[i*3+1]*3],r21);    
        cross(r10, r21,S);
        area[i] = norm(S)/2;
        //printf("Thread %d:\tArea %f\n",i,area[i]);
    }
    else {
        area[i] = 0;
    }
}
__global__ void volumeKernel(double * volume, double * vert, unsigned int * facets, unsigned int numFacets){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    double s01[3];
    if (i < numFacets){
        cross(&vert[facets[i*3]*3], &vert[facets[i*3+1]*3],s01);
        volume[i] = abs(dot(s01,&vert[facets[i*3+2]*3]))/6;
    }
    else {
        volume[i] = 0;
    }

}
__global__ void addTree(double* g_idata, double* g_odata){
    //https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

    extern __shared__ double sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x; // get the id of this thread
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    
        //printf("tid: %d\ti:%d\ti + blockDim.x:%d\tg_idata[i]:%f\tg_idata[i + blockDim.x]%f\n",tid,i, i + blockDim.x, g_idata[i] , g_idata[i + blockDim.x]);
        sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];// g_idata[i]; // move the data over
        g_idata[i] = 0;
        g_idata[i + blockDim.x] = 0;
        // printf("tid: %d\ti:%d\ti + blockDim.x:%d\tg_idata[i]:%f\tg_idata[i + blockDim.x]%f\t sdata[tid]: %f\n", tid, i, i + blockDim.x, g_idata[i], g_idata[i + blockDim.x], sdata[tid]);
                                                        //}
    __syncthreads();
            // do reduction in shared mem
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                sdata[tid] += sdata[tid + s];
            }
        __syncthreads();
        }
        __syncthreads();
        
        //     write result for this block to global mem
        if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

// template <unsigned int blockSize> __device__ void warpReduce(volatile double *sdata, unsigned int tid) {
//         if (blockSize >=  64) sdata[tid] += sdata[tid + 32];
//         if (blockSize >=  32) sdata[tid] += sdata[tid + 16];
//         if (blockSize >=  16) sdata[tid] += sdata[tid +  8];
//         if (blockSize >=    8) sdata[tid] += sdata[tid +  4];
//         if (blockSize >=    4) sdata[tid] += sdata[tid +  2];
//         if (blockSize >=    2) sdata[tid] += sdata[tid +  1];
// }
// template <unsigned int blockSize> __global__ void reduce6(double *g_idata,double *g_odata, unsigned int n) {
//     extern __shared__ double sdata[];
//     unsigned int tid = threadIdx.x;
//     unsigned int i = blockIdx.x*(blockSize*2) + tid;
//     unsigned int gridSize = blockSize*2*gridDim.x;
//     sdata[tid] = 0;
//     while (i < n) {
//         sdata[tid] += g_idata[i] + g_idata[i+blockSize];
//         i += gridSize;
//     }
//     __syncthreads();
//     if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
//     if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
//     if (blockSize >= 128) { if (tid <   64) { sdata[tid] += sdata[tid +   64]; } __syncthreads(); }
//     if (tid < 32) warpReduce(sdata, tid);
//     if (tid == 0) g_odata[blockIdx.x] = sdata[0];
// }

__global__ void addWithMultKernel(double *a ,double *b,double lambda, unsigned int size){
    // a += b * lambda
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<size){
        *(a+i) += *(b+i) * lambda;
    }
}

__global__ void areaGradient(double* gradAFacet, unsigned int* facets,double* verts,unsigned int numFacets){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    double S0[3];
    double S1[3];
    double S01[3];
    double S010[3];
    double S011[3];
    if (i<numFacets){
        vectorSub(&verts[facets[i*3+1]*3], &verts[facets[i*3]*3],S0);
        vectorSub(&verts[facets[i*3+2]*3], &verts[facets[i*3+1]*3],S1);
        cross(S0,S1,S01);
        cross(S01,S0,S010);
        cross(S01,S1,S011);
        // each facet has 3 vertices with gradient each, so in total 9 numbers we write them down here;
        
        // or facet i this is the gradent vector for its 0th vertex 
        vecAssign(&gradAFacet[i*9],S011,1.0/(2 * norm(S01)));

        // reuse S0 
        vectorAdd(S011,S010,S0);
        vecAssign(&gradAFacet[i*9 + 3],S0,-1.0/(2 * norm(S01)));

        vecAssign(&gradAFacet[i*9 + 6],S010,1.0/(2 * norm(S01)));
    }

}

__global__ void areaGradient2(double* gradAVert, unsigned int* facets,double* verts, unsigned int numFacets){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    double S0[3];
    double S1[3];
    double S01[3];
    double S010[3];
    double S011[3];

    if (i<numFacets){
        vectorSub(&verts[facets[i*3+1]*3], &verts[facets[i*3]*3],S0);
        vectorSub(&verts[facets[i*3+2]*3], &verts[facets[i*3+1]*3],S1);
        cross(S0,S1,S01);
        cross(S01,S0,S010);
        cross(S01,S1,S011);
        // each facet has 3 vertices with gradient each, so in total 9 numbers we write them down here;
        
        // or facet i this is the gradent vector for its 0th vertex 
        vectorAdd(S011,S010,S0);
		double Norm2inv = 1.0/(2.0*norm(S01));
		for (int j=0; j<3;j++){
			atomic_Add(&gradAVert[facets[i*3]*3+j],S011[j]*Norm2inv);
			atomic_Add(&gradAVert[facets[i*3+1]*3+j],-S0[j]*Norm2inv);
			atomic_Add(&gradAVert[facets[i*3+2]*3+j],S010[j]*Norm2inv);
		}



				// 		gradAVert[facets[i*3+1]*3]  -=S0[0]*Norm2inv;
		// 		gradAVert[facets[i*3+1]*3+1]-=S0[1]*Norm2inv;
		// 		gradAVert[facets[i*3+1]*3+2]-=S0[2]*Norm2inv;

						// 		gradAVert[facets[i*3+2]*3]  +=S010[0]*Norm2inv;
		// 		gradAVert[facets[i*3+2]*3+1]+=S010[1]*Norm2inv;
		// 		gradAVert[facets[i*3+2]*3+2]+=S010[2]*Norm2inv;



		// for (int ii = 0; ii<=maxAdd; ii++){
		// 	 __syncthreads(); // problem is that this is a local sync got a global one, we really should be doing a global one here
		// 	if (addOrder[i*3]==ii){
				
		// 		gradAVert[facets[i*3]*3]  +=S011[0]*Norm2inv;
		// 		gradAVert[facets[i*3]*3+1]+=S011[1]*Norm2inv;
		// 		gradAVert[facets[i*3]*3+2]+=S011[2]*Norm2inv;
		// 		if (facets[i*3]==VERTTOCHECK){
		// 			printf("adding {%f,%f,%f} from facet %d i am %d in the add order\t",S011[0]*Norm2inv,S011[1]*Norm2inv,S011[2]*Norm2inv,i,addOrder[i*3]);
		// 			printf("got {%f,%f,%f}\n",gradAVert[0+3*VERTTOCHECK],gradAVert[1+3*VERTTOCHECK],gradAVert[2+3*VERTTOCHECK]);
		// 		}
				
		// 	}
		// 	if (addOrder[i*3+1]==ii){
		// 		gradAVert[facets[i*3+1]*3]  -=S0[0]*Norm2inv;
		// 		gradAVert[facets[i*3+1]*3+1]-=S0[1]*Norm2inv;
		// 		gradAVert[facets[i*3+1]*3+2]-=S0[2]*Norm2inv;
		// 		if (facets[i*3+1]==VERTTOCHECK){
		// 			printf("adding {%f,%f,%f} from facet %d i am %d in the add order\t",S0[0]*Norm2inv,S0[1]*Norm2inv,S0[2]*Norm2inv,i,addOrder[i*3+2]);
		// 			printf("got {%f,%f,%f}\n",gradAVert[0+3*VERTTOCHECK],gradAVert[1+3*VERTTOCHECK],gradAVert[2+3*VERTTOCHECK]);
		// 		}
		// 	}
		// 	if (addOrder[i*3+2]==ii){
		// 		gradAVert[facets[i*3+2]*3]  +=S010[0]*Norm2inv;
		// 		gradAVert[facets[i*3+2]*3+1]+=S010[1]*Norm2inv;
		// 		gradAVert[facets[i*3+2]*3+2]+=S010[2]*Norm2inv;
		// 		if (facets[i*3+2]==VERTTOCHECK){
		// 			printf("adding {%f,%f,%f} from facet %d i am %d in the add order\t",S010[0]*Norm2inv,S010[1]*Norm2inv,S010[2]*Norm2inv,i,addOrder[i*3+2]);
		// 			printf("got {%f,%f,%f}\n",gradAVert[0+3*VERTTOCHECK],gradAVert[1+3*VERTTOCHECK],gradAVert[2+3*VERTTOCHECK]);
		// 		}
		// 	}
		// }
    }

}
__global__ void areaGradient3(double* gradAVert, unsigned int* facets,double* verts, unsigned int numFacets){
    int i = blockDim.x * blockIdx.x + threadIdx.x; //which facet is calculating
	//int j = threadIdx.y; // which vertex on this facet are we calculating for
	int k = threadIdx.y; // which (x,y,z) compoent of the vertex are we calculating
    
	extern __shared__ double sdata[]; // need 5x3xblocksize double for the whole block; I.E each thread in the block needs 5 3 vectors
    double *S0 = sdata+5*3*threadIdx.x;
    double *S1 = S0+3;
    double *S01 = S1+3;
    double *S010 = S01+3;
    double *S011 = S010+3;

    if (i<numFacets){
		S0[k] = verts[facets[i*3+1]*3+k] - verts[facets[i*3]*3+k];
		S1[k] = verts[facets[i*3+2]*3+k] - verts[facets[i*3+1]*3+k];
		__syncthreads();

		cross3(S0,S1,S01,k);
		__syncthreads();
        
		cross3(S01,S0,S010,k);
        cross3(S01,S1,S011,k);
		__syncthreads();
        S0[k] = S011[k]+S010[k];
        __syncthreads();
		// hmmm this is pretty in effiecnt 
        // or facet i this is the gradent vector for its 0th vertex 
		double Norm2inv = 1.0/(2.0*norm(S01));
		atomic_Add(&gradAVert[facets[i*3]*3+k],S011[k]*Norm2inv);
		atomic_Add(&gradAVert[facets[i*3+1]*3+k],-S0[k]*Norm2inv);
		atomic_Add(&gradAVert[facets[i*3+2]*3+k],S010[k]*Norm2inv);
	}
}

__global__ void volumeGradient(double* gradVFacet, unsigned int* facets,double* verts,unsigned int numFacets){
    // TO DO: this can this can be broken up into 3 for even faster computaiton
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    double c[3];
    double s = 1;
    if (i<numFacets){
        cross(&verts[facets[i*3]*3],&verts[facets[i*3+1]*3],c);
        s = sign(dot(c,&verts[facets[i*3+2]*3]));

        cross(&verts[facets[i*3+1]*3],&verts[facets[i*3+2]*3],c);
        vecAssign(&gradVFacet[i*9],c,s/6);

        cross(&verts[facets[i*3+2]*3],&verts[facets[i*3]*3],c);
        vecAssign(&gradVFacet[i*9 + 3],c,s/6);

        cross(&verts[facets[i*3]*3],&verts[facets[i*3+1]*3],c);
        vecAssign(&gradVFacet[i*9 + 6],c,s/6);
    }

}

__global__ void volumeGradient2(double* gradVVert, unsigned int* facets,double* verts,unsigned int numFacets){
    // TO DO: this can this can be broken up into 3 for even faster computaiton
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    double a[3];
	double b[3];
	double c[3];
    double s = 1;
    if (i<numFacets){
        cross(&verts[facets[i*3]*3],&verts[facets[i*3+1]*3],c);
        s = sign(dot(c,&verts[facets[i*3+2]*3]));
		//printf("facet %d, s = %f, c= {%f,%f,%f}\n",i,s,c[0],c[1],c[2]);
        cross(&verts[facets[i*3+1]*3],&verts[facets[i*3+2]*3],a);
        cross(&verts[facets[i*3+2]*3],&verts[facets[i*3]*3],b);
        cross(&verts[facets[i*3]*3],&verts[facets[i*3+1]*3],c);

		for (int j=0; j<3;j++){
			atomic_Add(&gradVVert[facets[i*3]*3+j],a[j]*s/6);
			// if (a[j]*s/6>1e-6){
			// printf("I am Facet %d, place %d, component %d and my term is %f I have added to vertex %d it is now %f\n",i,0,j,a[j]*s/6,facets[i*3],gradVVert[facets[i*3]*3+j]);
			// }

			atomic_Add(&gradVVert[facets[i*3+1]*3+j],b[j]*s/6);
			// if (b[j]*s/6>1e-6){
			// printf("I am Facet %d, place %d, component %d and my term is %f I have added to vertex %d it is now %f\n",i,1,j,b[j]*s/6,facets[i*3+1],gradVVert[facets[i*3+1]*3+j]);
			// }

			atomic_Add(&gradVVert[facets[i*3+2]*3+j],c[j]*s/6);
			// if (c[j]*s/6>1e-6){
			// printf("I am Facet %d, place %d, component %d and my term is %f I have added to vertex %d it is now %f\n",i,2,j,c[j]*s/6,facets[i*3+2],gradVVert[facets[i*3+2]*3+j]);
			// }

		}
    }

}
__global__ void volumeGradient3(double* gradVVert, unsigned int* facets,double* verts,unsigned int numFacets){
    // TO DO: this can this can be broken up into 3 for even faster computaiton
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = threadIdx.y; // which vertex on this facet are we calculating for
	int k = threadIdx.z; // which (x,y,z) compoent of the vertex are we calculating
	extern __shared__ double sdata[]; // just one piece of data per facet
	// lets load the 3 vectors into shared data frist so now we need 10 data per facet
	if (j ==0 && k ==0) sdata[threadIdx.x] = 0;

	__syncthreads();


	double term; 
    
	double s = 1;



    if (i<numFacets){
		if (j==0){
        	term = cross3oneComp(&verts[facets[i*3]*3],&verts[facets[i*3+1]*3],k);
			// now take the sign of c dotted with &verts[facets[i*3+2]*3]
			atomic_Add(&sdata[threadIdx.x],term*verts[facets[i*3+2]*3+k]);
		}
	}
	__syncthreads();

    if (i<numFacets){

        s = sign(sdata[threadIdx.x]);
		term = cross3oneComp(&verts[facets[i*3+(j+1)%3]*3],&verts[facets[i*3+(j+2)%3]*3],k);

		atomic_Add(&gradVVert[facets[i*3+j]*3+k],term*s/6.0);
	}
}
		

__global__ void facetToVertex(double* vertexValue, double* facetValue,unsigned int* vertToFacet, unsigned int* vertIndexStart,unsigned int numVert){
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = threadIdx.y;

    if (i<numVert){
        //first set to 0
        vertexValue[i*3 + j] = 0;
        for (int index = vertIndexStart[i]; index < vertIndexStart[i+1]; index++){
			vertexValue[i*3+j]+=facetValue[3*vertToFacet[index]+j];
            //vectorAdd(&vertexValue[i*3],&facetValue[3*vertToFacet[index]],&vertexValue[i*3]);
            //printf("vertex %d gets [%f,%f,%f]\n",i,facetValue[3*vertToFacet[index]],facetValue[3*vertToFacet[index]+1],facetValue[3*vertToFacet[index]+2]);
        }
    }
}

__global__ void projectForce(double* force,double* gradAVert,double* gradVVert,double scale,unsigned int numEle){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<numEle){
        force[i] = - (gradAVert[i] - scale * gradVVert[i]);
    }
}

__global__ void elementMultiply(double* v1, double* v2, double* out, unsigned int size){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<size){
        out[i] = v1[i]*v2[i];
        //printf("Thread %d: out value %f\n",i,out[i]);
    }
}


