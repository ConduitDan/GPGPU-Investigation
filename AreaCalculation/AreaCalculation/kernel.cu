#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>

#include <stdlib.h>
#include <stdbool.h>  
#include <string.h>

#define BLOCKSIZE 2
int getNumVertices(FILE* fp);
int getNumFacets(FILE* fp);
bool readInMesh(const char* fileName, float* verts, unsigned int* facets, unsigned int* nVert, unsigned int* nFace);

hipError_t areaWithCuda(float* vertices, unsigned int  meshSize, unsigned int* facets, \
    unsigned int facetSize, float* areaPerFace, float* area);

__global__ void areaKernel(float *area, const float *vertices, const unsigned int * facets, const int size)
{
    // given a set of vertices and facet [v0,v1,v2](list of indeices of vertices belonging to a face) fill in what the area of that face is
    
    // formula is (x1*y2+x2*y3+x3*y1-y1*x2-y2*x3-y3*x1)/2 
    // NOTE THIS CAN BE DONE MORE IN PARALLEL
    // Check for vetorized instruction for cross product

    
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // do i*3 because we have 3 vertcies per facet
    // do facets[]*2 becasue we have x and y positions
    if (i < size) {
        area[i] = abs(vertices[facets[i * 3] * 2] * (vertices[facets[i * 3 + 1] * 2 + 1] - vertices[facets[i * 3 + 2] * 2 + 1]) \
            + vertices[facets[i * 3 + 1] * 2] * (vertices[facets[i * 3 + 2] * 2 + 1] - vertices[facets[i * 3] * 2 + 1]) \
            + vertices[facets[i * 3 + 2] * 2] * (vertices[facets[i * 3] * 2 + 1] - vertices[facets[i * 3 + 1] * 2 + 1])) / 2;
    }
    else {
        area[i] = 0;
    }
}

__global__ void areaKernel3d(float* area, const float* vertices, const unsigned int* facets, const int size)
{
    // given a set of vertices and facet [v0,v1,v2](list of indeices of vertices belonging to a face) fill in what the area of that face is

    // formula is (x1*y2+x2*y3+x3*y1-y1*x2-y2*x3-y3*x1)/2 
    // NOTE THIS CAN BE DONE MORE IN PARALLEL
    // Check for vetorized instruction for cross product


    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // do i*3 because we have 3 vertcies per facet
    // do facets[]*2 becasue we have x and y positions
    float dx1 = vertices[facets[i * 3] * 3] - vertices[facets[i * 3 + 1] * 3];
    float dx2 = vertices[facets[i * 3 + 1] * 3] - vertices[facets[i * 3 + 2] * 3];
    float dy1 = vertices[facets[i * 3] * 3 + 1] - vertices[facets[i * 3 + 1] * 3 + 1];
    float dy2 = vertices[facets[i * 3 + 1] * 3 + 1] - vertices[facets[i * 3 + 2] * 3 + 1];
    float dz1 = vertices[facets[i * 3] * 3 + 2] - vertices[facets[i * 3 + 1] * 3 + 2];
    float dz2 = vertices[facets[i * 3 + 1] * 3 + 2] - vertices[facets[i * 3 + 2] * 3 + 2];
    if (i < size) {
        area[i] = abs(dx1*(dy2-dz2)+dx2*(dz1-dy1) + dy1*dz2 - dz1*dy2)/2;
    }
    else {
        area[i] = 0;
    }
}


__global__ void addTree(const float* g_idata, float* g_odata)
{
    //https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x; // get the id of this thread
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    //if (i + blockDim.x < size) {
    //printf("tid: %d\ti:%d\ti + blockDim.x:%d\tg_idata[i]:%f\tg_idata[i + blockDim.x]%f\n",tid,i, i + blockDim.x, g_idata[i] , g_idata[i + blockDim.x]);
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];// g_idata[i]; // move the data over
    //printf("tid: %d\ti:%d\ti + blockDim.x:%d\tg_idata[i]:%f\tg_idata[i + blockDim.x]%f\t sdata[tid]: %f\n", tid, i, i + blockDim.x, g_idata[i], g_idata[i + blockDim.x], sdata[tid]);
                                                      //}
   __syncthreads();
        // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
       __syncthreads();
    }
    __syncthreads();
    
//     write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];

 }

int main()
{

    // read in mesh

    unsigned int meshSize = 0;
    unsigned int facetSize = 0;


    /*const float vertices[meshSize * 2] = {\
        0,0,\
        1,0,\
        2,0,\
        0.5,1,\
        1.5,1,\
        0,2,\
        1,2,\
        2,2 };
    //x---x---x
    // \  /\  /
    //  \/  \/
    //  x----x
    //  /\   /\ 
    // /  \ /  \
    //x----x----x*/
    float* vertices = NULL;

    /*const unsigned int facets[facetSize * 3] = {0, 1, 3, \
                                        3, 4, 1, \
                                        1, 2, 4, \
                                        3, 5, 6, \
                                        3, 6, 4, \
                                        6, 4, 7 };
    */
    unsigned int * facets = NULL;
    bool readSuccess = readInMesh("sphere.mesh", vertices, facets, &meshSize, &facetSize);
    if (!readSuccess) {
        fprintf(stderr, "failed to read in mesh");
        return -1;
    }
    fprintf(stdout, "Read in mesh with %d vertices and %d faces\n", meshSize, facetSize);


    float *areaPerFace = (float *) malloc(facetSize * sizeof(float));
    float area = 0;
    float areaCPU = 0;
    // Add vectors in parallel.
    hipError_t cudaStatus = areaWithCuda(vertices, meshSize, facets, facetSize, areaPerFace, &area);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "areaWithCuda failed!");
        return 1;
    }

    printf("Facets:\n");
    for (int i = 0; i < facetSize; i++) {
        printf("%d: (%f, %f), (%f, %f) , (%f, %f)\n", i, vertices[facets[i * 3] * 2], vertices[facets[i * 3] * 2 + 1],\
            vertices[facets[i * 3 + 1] * 2], vertices[facets[i * 3 + 1] * 2 + 1],\
            vertices[facets[i * 3 + 2] * 2], vertices[facets[i * 3 + 2] * 2 + 1]);
    }

    printf("area on facets: \n");
    for (int i = 0; i < facetSize; i++) {
        //areaCPU = (vertices[facets[i * 3] * 2] * (vertices[facets[i * 3 + 1] * 2 + 1] - vertices[facets[i * 3 + 2] * 2 + 1]) \
            + vertices[facets[i * 3 + 1] * 2] * (vertices[facets[i * 3 + 2] * 2 + 1] - vertices[facets[i * 3] * 2 + 1]) \
            + vertices[facets[i * 3 + 2] * 2] * (vertices[facets[i * 3] * 2 + 1] - vertices[facets[i * 3 + 1] * 2 + 1])) / 2;
        printf("%d: GPU %f \n", i, areaPerFace[i]);
    }
    printf("Total Area: %f\n", area);


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t areaWithCuda(float* vertices, unsigned int  meshSize, unsigned int* facets, \
    unsigned int facetSize, float * areaPerFace, float * area)
{
    float *dev_vertices = 0;
    unsigned int *dev_facets = 0;
    float *dev_areaPerFace = 0;
    float *dev_areaSum = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    unsigned int BufferedSize = ceil(facetSize / (float)(2 * BLOCKSIZE)) * 2 * BLOCKSIZE;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    // round up areaPerFace so that every thread in every block can assign and do something
    cudaStatus = hipMalloc((void**)&dev_areaPerFace, BufferedSize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    unsigned int numSum = BufferedSize / BLOCKSIZE / 2;
    cudaStatus = hipMalloc((void**)&dev_areaSum, numSum * sizeof(float)); // this should be facetSize/Num
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_vertices, 2 * meshSize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_facets, 3 * facetSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_vertices, vertices, 3 * meshSize * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! vertices\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_facets, facets, 3 * facetSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! facets\n");
        goto Error;
    }


    unsigned int areaNumBlock = ceil(BufferedSize / (float)BLOCKSIZE);

    // Launch a kernel on the GPU with one thread for each element.
    areaKernel3d <<<areaNumBlock, BLOCKSIZE>>> (dev_areaPerFace, dev_vertices, dev_facets, facetSize);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "areaKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error %s after launching areaKernel!\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    unsigned int addNumBlock = ceil(BufferedSize / (float)BLOCKSIZE / 2.0);
    // now sum the result
    addTree << <addNumBlock, BLOCKSIZE, BufferedSize / 2 * sizeof(float) >> > (dev_areaPerFace, dev_areaSum);
    for (int i = addNumBlock; i > 1; i /= (BLOCKSIZE * 2)) {
        addTree << <ceil((float)addNumBlock/ (BLOCKSIZE * 2)), BLOCKSIZE, BufferedSize / 2 * sizeof(float) >> > (dev_areaSum, dev_areaSum);
    }
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error %s after launching addKernel!\n", hipGetErrorString(cudaStatus));
        goto Error;
    }






    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(areaPerFace, dev_areaPerFace, facetSize * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! area per facet\n");
        goto Error;
    }
    cudaStatus = hipMemcpy(area, dev_areaSum,sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! area\n");
        goto Error;
    }


Error:


    hipFree(dev_vertices);
    hipFree(dev_facets);
    hipFree(dev_areaPerFace);
    hipFree(dev_areaSum);

    return cudaStatus;
}

int getNumVertices(FILE* fp) {

    int numVertices = 0;
    char line[50];
    fscanf(fp, "%s\n", line);
    if (strcmp(line,"vertices")) {
        fprintf(stderr, "File didn't start with 'vertices'\n");
        return -1;

    }
    fgets(line, 50, fp); // eat the new line
    fgets(line, 50, fp); // read line 1
    numVertices++;
    while (strcmp(line ,"\n")&& !feof(fp)) {
        numVertices++;
        fgets(line, 50, fp); // read line n
    }
return numVertices;

}
int getNumFacets(FILE* fp) {
    int numFaces = 0;
    char line[50];
    fscanf(fp, "%s\n", line);
    while (strcmp(line,"faces")) {
        fscanf(fp, "%s\n", line);
        if (feof(fp)) {
            fprintf(stderr, "File had no faces\n");
            return -1;
        }
    }
    fgets(line, 50, fp); // eat the new line
    fgets(line, 50, fp); // read line 1
    numFaces++;
    while (strcmp(line,"\n")&& !feof(fp)) {
        numFaces++;
        fgets(line, 50, fp); // read line 1

    }
    return numFaces;

}


bool readInMesh(const char* fileName, float* verts, unsigned int* facets, unsigned int* nVert, unsigned int* nFace) {
    FILE* fp;
    char* line = NULL;
    size_t len = 0;
    char sectionHeader[50];

    int numAssigned = 0;

    fp = fopen(fileName, "r");
    if (fp == NULL) {
        fprintf(stderr, "Could not open file\n");
            return false;
    }

    *nVert = getNumVertices(fp);
    *nFace = getNumFacets(fp);

    verts = (float*)malloc(*nVert * 3 * sizeof(float)); // [x0; y0; z0; x1; y1;.... ]
    facets = (unsigned int*)malloc(*nFace * 3 * sizeof(unsigned int));// [a0; b0; c0; a1;b1;c1;...]


    rewind(fp); // rewind the file to the beginning
    // make sure the first line say vertices

    fscanf(fp, "%s\n", sectionHeader);
    if (strcmp(sectionHeader ,"vertices")) {
        fprintf(stderr, "File didn't start with 'vertices'\n");
        return false;
    }
    // get past the empty line

    float tmp0, tmp1, tmp2;
    for (int i = 0; i < *nVert; i++) {
        numAssigned = fscanf(fp, "%*d %f %f %f\n", &tmp0, &tmp1, &tmp2);
 
        if (numAssigned < 3) {
            fprintf(stderr, "bad file format\n");
            return false;
        }
        verts[i * 3] = (float) tmp0;
        verts[i * 3 + 1] = (float) tmp1;
        verts[i * 3 + 2] = (float) tmp2;

    }

    fscanf(fp, "%*d");
    fscanf(fp, "%s\n", sectionHeader);
    while (strcmp(sectionHeader ,"faces")) {
        fscanf(fp, "%s\n", sectionHeader);
        if (feof(fp)) {
            fprintf(stderr, "couldn't find faces\n");
            return false;
        }
    }

    for (int i = 0; i < *nFace; i++) {
        numAssigned = fscanf(fp, "%*d %d %d %d\n", &facets[i * 3], &facets[i * 3 + 1], &facets[i * 3 + 2]);
        if (numAssigned < 3) {
            fprintf(stderr, "bad file format for faces\n");
            return false;
        }

    }
    return true;

}
