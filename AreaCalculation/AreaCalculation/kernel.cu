#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>

#define BLOCKSIZE 2

hipError_t areaWithCuda(const float* vertices, const unsigned int  meshSize, const unsigned int* facets, \
    const unsigned int facetSize, float* areaPerFace, float* area);

__global__ void areaKernel(float *area, const float *vertices, const unsigned int * facets, const int size)
{
    // given a set of vertices and facet [v0,v1,v2](list of indeices of vertices belonging to a face) fill in what the area of that face is
    
    // formula is (x1*y2+x2*y3+x3*y1-y1*x2-y2*x3-y3*x1)/2 
    // NOTE THIS CAN BE DONE MORE IN PARALLEL
    // Check for vetorized instruction for cross product

    
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // do i*3 because we have 3 vertcies per facet
    // do facets[]*2 becasue we have x and y positions
    if (i < size) {
        area[i] = abs(vertices[facets[i * 3] * 2] * (vertices[facets[i * 3 + 1] * 2 + 1] - vertices[facets[i * 3 + 2] * 2 + 1]) \
            + vertices[facets[i * 3 + 1] * 2] * (vertices[facets[i * 3 + 2] * 2 + 1] - vertices[facets[i * 3] * 2 + 1]) \
            + vertices[facets[i * 3 + 2] * 2] * (vertices[facets[i * 3] * 2 + 1] - vertices[facets[i * 3 + 1] * 2 + 1])) / 2;
    }
    else {
        area[i] = 0;
    }
}


__global__ void addTree(const float* g_idata, float* g_odata)
{
    //https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x; // get the id of this thread
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    //if (i + blockDim.x < size) {
    //printf("tid: %d\ti:%d\ti + blockDim.x:%d\tg_idata[i]:%f\tg_idata[i + blockDim.x]%f\n",tid,i, i + blockDim.x, g_idata[i] , g_idata[i + blockDim.x]);
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];// g_idata[i]; // move the data over
    //printf("tid: %d\ti:%d\ti + blockDim.x:%d\tg_idata[i]:%f\tg_idata[i + blockDim.x]%f\t sdata[tid]: %f\n", tid, i, i + blockDim.x, g_idata[i], g_idata[i + blockDim.x], sdata[tid]);
                                                      //}
   __syncthreads();
        // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
       __syncthreads();
    }
    __syncthreads();
    
//     write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];

 }

int main()
{
    const int meshSize = 8;
    const int facetSize = 6;
    const float vertices[meshSize * 2] = {\
        0,0,\
        1,0,\
        2,0,\
        0.5,1,\
        1.5,1,\
        0,2,\
        1,2,\
        2,2 };
    //x---x---x
    // \  /\  /
    //  \/  \/
    //  x----x
    //  /\   /\ 
    // /  \ /  \
    //x----x----x
    const unsigned int facets[facetSize * 3] = { 0, 1, 3, \
                                        3, 4, 1, \
                                        1, 2, 4, \
                                        3, 5, 6, \
                                        3, 6, 4, \
                                        6, 4, 7 };
    float *areaPerFace = (float *) malloc(facetSize * sizeof(float));
    float area = 0;
    float areaCPU = 0;
    // Add vectors in parallel.
    hipError_t cudaStatus = areaWithCuda(vertices, meshSize, facets, facetSize, areaPerFace, &area);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "areaWithCuda failed!");
        return 1;
    }

    printf("Facets:\n");
    for (int i = 0; i < facetSize; i++) {
        printf("%d: (%f, %f), (%f, %f) , (%f, %f)\n", i, vertices[facets[i * 3] * 2], vertices[facets[i * 3] * 2 + 1],\
            vertices[facets[i * 3 + 1] * 2], vertices[facets[i * 3 + 1] * 2 + 1],\
            vertices[facets[i * 3 + 2] * 2], vertices[facets[i * 3 + 2] * 2 + 1]);
    }

    printf("area on facets: \n");
    for (int i = 0; i < facetSize; i++) {
        //areaCPU = (vertices[facets[i * 3] * 2] * (vertices[facets[i * 3 + 1] * 2 + 1] - vertices[facets[i * 3 + 2] * 2 + 1]) \
            + vertices[facets[i * 3 + 1] * 2] * (vertices[facets[i * 3 + 2] * 2 + 1] - vertices[facets[i * 3] * 2 + 1]) \
            + vertices[facets[i * 3 + 2] * 2] * (vertices[facets[i * 3] * 2 + 1] - vertices[facets[i * 3 + 1] * 2 + 1])) / 2;
        printf("%d: GPU %f \n", i, areaPerFace[i]);
    }
    printf("Total Area: %f\n", area);


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t areaWithCuda(const float* vertices, const unsigned int  meshSize, const unsigned int* facets, \
    const unsigned int facetSize, float * areaPerFace, float * area)
{
    float *dev_vertices = 0;
    unsigned int *dev_facets = 0;
    float *dev_areaPerFace = 0;
    float *dev_areaSum = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    unsigned int BufferedSize = ceil(facetSize / (double)(2 * BLOCKSIZE)) * 2 * BLOCKSIZE;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    // round up areaPerFace so that every thread in every block can assign and do something
    cudaStatus = hipMalloc((void**)&dev_areaPerFace, BufferedSize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    unsigned int numSum = BufferedSize / BLOCKSIZE / 2;
    cudaStatus = hipMalloc((void**)&dev_areaSum, numSum * sizeof(float)); // this should be facetSize/Num
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_vertices, 2 * meshSize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_facets, 3 * facetSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_vertices, vertices, 2 * meshSize * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! vertices");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_facets, facets, 3 * facetSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! facets");
        goto Error;
    }


    unsigned int areaNumBlock = ceil(BufferedSize / (double)BLOCKSIZE);

    // Launch a kernel on the GPU with one thread for each element.
    areaKernel <<<areaNumBlock, BLOCKSIZE>>> (dev_areaPerFace, dev_vertices, dev_facets, facetSize);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "areaKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error %s after launching areaKernel!\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    unsigned int addNumBlock = ceil(BufferedSize / (double)BLOCKSIZE / 2.0);
    // now sum the result
    addTree << <addNumBlock, BLOCKSIZE, BufferedSize / 2 * sizeof(float) >> > (dev_areaPerFace, dev_areaSum);
    for (int i = addNumBlock; i > 1; i /= (BLOCKSIZE * 2)) {
        addTree << <ceil((double)addNumBlock/ (BLOCKSIZE * 2)), BLOCKSIZE, BufferedSize / 2 * sizeof(float) >> > (dev_areaSum, dev_areaSum);
    }
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error %s after launching addKernel!\n", hipGetErrorString(cudaStatus));
        goto Error;
    }






    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(areaPerFace, dev_areaPerFace, facetSize * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! area per facet\n");
        goto Error;
    }
    cudaStatus = hipMemcpy(area, dev_areaSum,sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! area\n");
        goto Error;
    }


Error:


    hipFree(dev_vertices);
    hipFree(dev_facets);
    hipFree(dev_areaPerFace);
    hipFree(dev_areaSum);

    return cudaStatus;
}
